#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define MM 0.001f
#define DIM 3
#define PI 3.1415927f
#define X 0
#define Y 1
#define Z 2
#define Proton_Charge_Quantity 1.6021766208e-19f
#define Proton_Static_MassKg 1.672621898e-27f
#define Proton_Static_MassMeV 938.2720813f
#define Light_Speed 299792458.0f
#define RUN_STEP 0.001f

__device__ __forceinline__ void add3d(float* a, float* b, float* ret)
{
    ret[X] = a[X] + b[X];
    ret[Y] = a[Y] + b[Y];
    ret[Z] = a[Z] + b[Z];
}

__device__ __forceinline__ void add3d_local(float* a_local, float* b)
{
    a_local[X] += b[X];
    a_local[Y] += b[Y];
    a_local[Z] += b[Z];
}

__device__ __forceinline__ void sub3d(float* a, float* b, float* ret)
{
    ret[X] = a[X] - b[X];
    ret[Y] = a[Y] - b[Y];
    ret[Z] = a[Z] - b[Z];
}

__device__ __forceinline__ void copy3d(float* src, float* des)
{
    des[X] = src[X];
    des[Y] = src[Y];
    des[Z] = src[Z];
}

__device__ __forceinline__ void cross3d(float* a, float* b, float* ret)
{
    ret[X] = a[Y] * b[Z] - a[Z] * b[Y];
    ret[Y] = -a[X] * b[Z] + a[Z] * b[X];
    ret[Z] = a[X] * b[Y] - a[Y] * b[X];
}

__device__ __forceinline__ void dot_a_v(float a, float* v)
{
    v[X] *= a;
    v[Y] *= a;
    v[Z] *= a;
}

__device__ __forceinline__ void dot_a_v_ret(float a, float* v, float* ret)
{
    ret[X] = v[X] * a;
    ret[Y] = v[Y] * a;
    ret[Z] = v[Z] * a;
}

__device__ __forceinline__ float dot_v_v(float* v1, float* v2)
{
    return v1[X] * v2[X] + v1[Y] * v2[Y] + v1[Z] * v2[Z];
}

__device__ __forceinline__ float len3d(float* v)
{
    return sqrtf(v[X] * v[X] + v[Y] * v[Y] + v[Z] * v[Z]);
}

__device__ __forceinline__ void neg3d(float* v)
{
    v[X] = -v[X];
    v[Y] = -v[Y];
    v[Z] = -v[Z];
}

// ע�⣬�������Ĳ��ǵ���Ԫ�Ĵų�������Ҫ���� ���� �� ��0/4�� (=1e-7)
// 2020��11��11�� ����ͨ��
__device__  void dB(float* p0, float* p1, float* p, float* ret)
{
    float p01[DIM];
    float r[DIM];
    float rr;

    sub3d(p1, p0, p01); // p01 = p1 - p0

    add3d(p0, p1, r); // r = p0 + p1

    dot_a_v(0.5, r); // r = (p0 + p1)/2

    sub3d(p, r, r); // r = p - r

    rr = len3d(r); // rr = len(r)

    cross3d(p01, r, ret); // ret = p01 x r

    rr = 1.0 / rr / rr / rr; // changed

    dot_a_v(rr, ret); // rr . (p01 x r)
}

// line = float[length][3]�����㵼�� line �� p ������Ĵų���length ��ʾ line �е���Ŀ������ֵ�ŵ� ret
__device__ void magnet_at_point(float** line, int length, float current, float* p, float* ret)
{
    int i;
    float db[3];

    ret[X] = 0.0f;
    ret[Y] = 0.0f;
    ret[Z] = 0.0f;

    for (i = 0; i < length - 1; i++)
    {
        dB(line[i], line[i + 1], p, db);
        add3d_local(ret, db);
    }

    dot_a_v(current * 1e-7, ret);
}

// ������һ�� m �ų���p λ�ã�v �ٶȣ�rm ��������sp ����
 __device__  __forceinline__  void particle_run_step(float* m, float* p, float* v, float run_mass, float speed)
{
    float a[3]; // ���ٶ�
    float t;    // �˶�ʱ��
    float d[3]; // λ�ñ仯 �ٶȱ仯

    // q v b
    cross3d(v, m, a); // a = v*b

    dot_a_v(Proton_Charge_Quantity / run_mass, a); // a = q v b / mass ���ٶ�

    t = RUN_STEP / speed; // �˶�ʱ��

    dot_a_v_ret(t, v, d); // d = t v λ�ñ仯

    add3d_local(p, d); // p+=d

    dot_a_v_ret(t, a, d); // d = t a �ٶȱ仯

    add3d_local(v, d); // v+=d
}

// �����ڵ����ߴų����˶������ڲ���
// len �˶����룬p λ�ã�v �ٶȣ�rm ��������sp ���ʣ�line[des_len][3] ���ߣ�des_len ���ߵ�����current ����
 __device__    void particle_run_len_one_line(float len, float* p, float* v, float run_mass, float speed, float** line, int des_len, float current)
{
    float distance = 0.0f;
    float m[3]; // �ų�
    while (distance < len)
    {
        magnet_at_point(line, des_len, current, p, m);
        particle_run_step(m, p, v, run_mass, speed);
        distance += RUN_STEP;
    }
}

 __device__  void linspace(float start, float end, int number, float* ret)
{
    int i;
    float d = (end - start) / (number - 1);

    for (i = 0; i < number - 1; i++)
    {
        ret[i] = start + d * i;
    }

    // ���һ������ֵ�����ټ������
    ret[number - 1] = end;
}

 // ���������������Դ�
 __global__    void test_solenoid() 
{
    int i;
    float t;
    float r = 0.01f;
    float length = 0.1f;
    int n = 20;
    int pas = 360;
    float total_theta = n * 2 * PI;

    float current = 10000.0f;

    int des_len = n * pas;

    float** lines = (float**)malloc(des_len * sizeof(float*));

    float* lins = (float*)malloc(des_len * sizeof(float));

    linspace(0, total_theta, des_len, lins);

    for (i = 0; i < des_len; i++)
    {
        lines[i] = (float*)malloc(DIM * sizeof(float));
        t = lins[i];
        lines[i][X] = cosf(t) * r;
        lines[i][Y] = sinf(t) * r;
        lines[i][Z] = t / total_theta * length;
    }

    float p[3] = { 0, 0, 0 };
    float v[3] = { 0.0, 0.0, 1.839551780274753E8 };

    particle_run_len_one_line(10.0, p, v, 2.1182873748205775E-27, 1.839551780274753E8, lines, des_len, current);

    printf("px=%f --cuda  ", p[X]);
    printf("py=%f --cuda ", p[Y]);
    printf("py=%f --cuda  ", p[Z]);
    printf("vx=%f --cuda  ", v[X]);
    printf("vy=%f --cuda  ", v[Y]);
    printf("vz=%f --cuda  ", v[Z]);

    free(lins);
    for (i = 0; i < des_len; i++)
    {
        free(lines[i]);
    }

    free(lines);
}

 // ���Ժ������� p �����߹ܴų�
 //part_deg 1�� 2�� 3�ȣ�������360Լ��
 __device__ void magnet_at_solenoid(float current,float small_r, float length, float number_wind, int part_deg, float* p,float* sin_table, float* ret) {
     int total_deg = 360 * number_wind;
     float m[3];
     
     int per_deg = 0;
     int cur_deg = part_deg;

     float pre_p[3];
     pre_p[X] = small_r * sin_table[90 + per_deg]; // sin(90+a)=cos(a)
     pre_p[Y] = small_r * sin_table[per_deg];
     pre_p[Z] = length * per_deg / 360.0 / number_wind;


     float cur_p[3];

     ret[X] = 0.0f;
     ret[Y] = 0.0f;
     ret[Z] = 0.0f;

     while (cur_deg <= total_deg) {
         cur_p[X] = small_r * sin_table[(90 + cur_deg)%360];
         cur_p[Y] = small_r * sin_table[cur_deg%360];
         cur_p[Z] = length * cur_deg / 360.0 / number_wind;

         // (float* p0, float* p1, float* p, float* ret)
         dB(pre_p, cur_p, p, m);
         
         add3d_local(ret, m);

         copy3d(cur_p, pre_p);

         cur_deg += part_deg;
     }

     dot_a_v(current * 1e-7, ret);
     
 }

 __global__ void magnet_at_solenoid_test_p000(float* sin_table) {

     float p[3] = { 0,0,0 };
     float m[3];
    
     magnet_at_solenoid(10000.0f, 0.01f, 0.1f, 20, 1, p, sin_table, m);

     printf("mx=%f --cuda  ", m[X]);
     printf("my=%f --cuda  ", m[Y]);
     printf("mz=%f --cuda  ", m[Z]);
 }

 __global__    void test_solenoid_no_malloc(float* sin_table)
 {
     float r = 0.01f;
     float length = 0.1f;
     int n = 20;
     float total_theta = n * 2 * PI;

     float current = 10000.0f;


     float run_len = 1.0f;

     float p[3] = { 0, 0, 0 };
     float v[3] = { 0.0, 0.0, 1.839551780274753E8 };
     float rm = 2.1182873748205775E-27;
     float speed = 1.839551780274753E8;
     float m[3];

     printf("px=%f --cuda  \n", p[X]);
     printf("py=%f --cuda \n", p[Y]);
     printf("py=%f --cuda  \n", p[Z]);
     printf("vx=%f --cuda  \n", v[X]);
     printf("vy=%f --cuda  \n", v[Y]);
     printf("vz=%f --cuda  \n", v[Z]);

     

     float distance = 0.0f;

     while (distance < run_len) {
         magnet_at_solenoid(current, r, length, n, 1, p, sin_table, m);
         particle_run_step(m, p, v, rm, speed);
         distance += RUN_STEP;
     }


     printf("px=%f --cuda  \n", p[X]);
     printf("py=%f --cuda \n", p[Y]);
     printf("py=%f --cuda  \n", p[Z]);
     printf("vx=%f --cuda  \n", v[X]);
     printf("vy=%f --cuda  \n", v[Y]);
     printf("vz=%f --cuda  \n", v[Z]);
 }




/**
 * Host main routine
 */
int
main(void)
{
    int i;
    float sin_table[360];
    for (i = 0; i < 360; i++) {
        sin_table[i] = sin(((float)i) / 180.0f * PI);
    }

    float* d_sin_tb;

    hipMalloc((void**)&d_sin_tb, 360 * sizeof(float));

    hipMemcpy(d_sin_tb, sin_table, 360 * sizeof(float), hipMemcpyHostToDevice);

    test_solenoid_no_malloc <<<256, 24>>>(d_sin_tb);

    hipDeviceSynchronize();

    hipFree(d_sin_tb);

    printf("Done\n");
    return 0;
}

